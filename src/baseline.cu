#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <thrust/device_vector.h>
#include "../include/baseline.cuh"

template <typename input_type>
std::vector<input_type> run_cpu_baseline(std::vector<input_type> const& left,
                                         std::vector<input_type> const& right,
                                         const int num_iterations, 
                                         bool record_time)
{
  std::vector<input_type> result{left};

  auto start = std::chrono::high_resolution_clock::now();

  for(int iter = 0; iter < num_iterations; ++iter)
  {
    for(int i = 0; i < left.size(); ++i)
      result[i] += right[i];
  }

  auto stop = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = stop - start;

  if(true == record_time)
  {
    std::cout << "CPU baseline elapsed time(s): " << elapsed.count() << "\n";
  }

  return result;
}

template std::vector<int> run_cpu_baseline<int>(std::vector<int> const& left,
                                                std::vector<int> const& right,
                                                const int num_iterations,
                                                bool record_time);

template std::vector<double> run_cpu_baseline<double>(std::vector<double> const& left,
                                                      std::vector<double> const& right,
                                                      const int num_iterations,
                                                      bool record_time); 

template <typename T>
__global__
void baseline_kernel(T * left, T * right, int64_t size){

  int64_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  while(idx < size)
  {
    left[idx] += right[idx];
    idx += blockDim.x * gridDim.x;
  }
}

template <typename input_type>
std::vector<input_type> run_gpu_baseline(std::vector<input_type> const& left,
                                         std::vector<input_type> const& right,
                                         const int num_iterations,
                                         bool record_time)
{

  thrust::device_vector<input_type> d_left{left};
  thrust::device_vector<input_type> d_right{right};

  constexpr int block_size = 256;
  const int grid_size = (left.size() + block_size - 1)/block_size;

  auto start = std::chrono::high_resolution_clock::now();

  for(int iter = 0; iter < num_iterations; ++iter)
  {
    baseline_kernel<<<grid_size, block_size>>>(d_left.data().get(), 
                                               d_right.data().get(),
                                               d_left.size());
  }

  if(hipSuccess != hipDeviceSynchronize()){
    std::cout << "GPU baseline failed!\n";
  }

  auto stop = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = stop - start;

  if(true == record_time)
  {
    std::cout << "gpu baseline elapsed time(s): " << elapsed.count() << "\n";
  }

  std::vector<input_type> result(left.size());
  hipMemcpy(result.data(), d_left.data().get(), d_left.size() * sizeof(input_type), hipMemcpyDeviceToHost);

  return result;

}

template std::vector<int> run_gpu_baseline<int>(std::vector<int> const& left,
                                                std::vector<int> const& right,
                                                const int num_iterations,
                                                bool record_time);
template std::vector<double> run_gpu_baseline<double>(std::vector<double> const& left,
                                                      std::vector<double> const& right,
                                                      const int num_iterations,
                                                      bool record_time);

